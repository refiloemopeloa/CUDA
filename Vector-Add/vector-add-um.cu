#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <stdio.h>

// CUDA Kernel for vector addition
// This function is executed by each and every thread, individually
__global__ void vectorAddUM(int* a, int* b, int* c, int n) {
    // Calculate global thread ID (tid)
              //which block we are in   //which thread we are in (offset in the block)
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
                          //block size
    // Vector boundary guard
    // Ensures that we dont go passed the number of elements in the vector, since it is possible that there are more threads than elements in the vector
    if (tid < n) {
        // Each thread adds a single element
        c[tid] = a[tid] + b[tid];
    }
}

// Initialize vector of size n to int between 0 and 99
void matrix_init(int* a, int n) {
    for (int i = 0; i < n; i++) {
        a[i] = rand() % 100;    }
}

// Check vector add result
void error_check(int *a, int* b, int *c, int n) {
    for (int i=0; i < n; i++) {
        assert(c[i] == a[i] + b[i]);
    }
}

int main() {
    int id = hipGetDevice(&id);

    // 2 ^ 16
    int n = 1 << 16;

    // Host vector pointers
//    int *h_a, *h_b, *h_c;

    // Device vector pointers
//    int *d_a, *d_b, *d_c;

    // Allocation size for all vectors
    size_t bytes = sizeof(int) * n;

    int *a, *b, *c;

    // Allocate host memory
    // a = (int *)malloc(bytes);
    // b = (int *)malloc(bytes);
    // c = (int *)malloc(bytes);

    // Allocate space on device memory for vectors
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Initialize vectors a and b with random values between 0 and 99
    matrix_init(a, n);
    matrix_init(b, n);

    // Copy data to from host memory to device memory
//    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
//    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Threadblock size
    int NUM_THREADS = 256;

    // Grid size - each grid will compute on n/NUM_THREADS of the vector
    int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

    // Launch kernel on default stream w/o shared memory

    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);

    vectorAddUM<<<NUM_BLOCKS, NUM_THREADS>>>(a,b,c,n);

    // Wait for all previous operations before using values
    hipDeviceSynchronize();

    // Copy sum vector from device to host
//    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

    // Check result for errors
    error_check(a, b, c, n);

    printf("Completed successfully\n");

    return 0;
}